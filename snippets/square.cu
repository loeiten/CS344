#include <stdio.h>
// NOTE: These are not needed if you compile with nvcc
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define ARRAY_SIZE (64)

__global__ void square(float* d_out, float* d_in) {
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f;
}

int main(int argc, char** argv) {
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // Generate the input array on the host
  float h_in[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; ++i) {
    h_in[i] = float(i);
  }
  float h_out[ARRAY_SIZE];

  // Declare GPU memory pointers
  float* d_in;
  float* d_out;

  // Allocate GPU memory
  hipMalloc((void**)&d_in, ARRAY_BYTES);
  hipMalloc((void**)&d_out, ARRAY_BYTES);

  // Transfer the data to the GPU
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // Launch the kernel
  square<<<1, ARRAY_SIZE>>>(d_out, d_in);

  // Copy back the result array to the CPU
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  // Print the resulting array
  for (int i = 0; i < ARRAY_SIZE; ++i) {
    printf("%f", h_out[i]);
    printf(((i % 4) != 3) ? "\t" : "\n");
  }

  // Free GPU memory allocation
  hipFree(d_in);
  hipFree(d_out);
}
